#include "hip/hip_runtime.h"
﻿#include "include_gpu/LevelSet_gpu.cuh"

//////////////////////////////////////////
///**** Device Function definitions ****/
/////////////////////////////////////////

__device__ __forceinline__ float norm(float3 a) {
    return sqrt(a.x * a.x + a.y * a.y + a.z * a.z);
}

__device__ __forceinline__ float3 make_ray(float x, float y, float z) {

    float3 ray = make_float3(x, y, z);
    ray = ray / sqrt(x * x + y * y + z * z);
    return ray;
}

__device__ __forceinline__ int IsInterectingRayTriangle3D_gpu(float3 ray, float3 p0, float3 p1, float3 p2, float3 p3, float3 n) { 
    float den = dot(ray, n);
    if (fabs(den) < 1.0e-6f) {
        if (dot(p1 - p0, n) == 0.0f) {
            return 0;
        }
        return 0;
    }

    float fact = (dot(p1 - p0, n) / den);
    if (fact < 1.0e-6f)
        return 0;

    float3 proj = p0 + ray * fact;
    // Compute if proj is inside the triangle
    // V = p1 + s(p2-p1) + t(p3-p1)
    // find s and t
    float3 u = p2 - p1;
    float3 v = p3 - p1;
    float3 w = proj - p1;

    float s = (dot(u, v) * dot(w, v) - dot(v, v) * dot(w, u)) / (dot(u, v) * dot(u, v) - dot(u, u) * dot(v, v));
    float t = (dot(u, v) * dot(w, u) - dot(u, u) * dot(w, v)) / (dot(u, v) * dot(u, v) - dot(u, u) * dot(v, v));


    if (s >= 0.0f && t >= 0.0f && s + t <= 1.0f) {
        float3 t_12 = cross(u, w);
        if (norm(t_12) < 1.0e-6f) {
            return 0;
        }

        float3 t_13 = cross(v, w);
        if (norm(t_13) < 1.0e-6f) {
            return 0;
        }

        float3 t_23 = cross((p3 - proj), (p3 - p2));
        if (norm(t_23) < 1.0e-6f) {
            return 0;
        }
        return 1;
    }

    //coordonnees barycentriques
    //float3 bary;

    // The area of a triangle is 
    //float areaABC = fabs(dot(n, cross((p1 - p2), (p3 - p2))));
    //float areaPBC = fabs(dot(n, cross((p1 - proj), (p3 - proj))));
    //float areaPCA = fabs(dot(n, cross((p3 - proj), (p2 - proj))));

    //bary.x = areaPBC / areaABC; // alpha
    //bary.y = areaPCA / areaABC; // beta
    //bary.z = 1.0f - bary.x - bary.y; // gamma
    //
    //if (bary.z < 0.0f && bary.z < 1.0f)
    //    return 1.0f;

    /*if (bary.z < 0.0f)
        return 0.0f;

    if ()*/
    /*if (fabs(t) < 1.0e-6f || fabs(s) < 1.0e-6f || fabs(s + t) < 1 + 1.0e-6f)
        return 0.5f;*/

    /*if (t == 0.0f && s >= 0.0f && s <= 1.0f)
        return 0.5f;

    if (s == 0.0f && t >= 0.0f && t <= 1.0f)
        return 0.5f;

    if (s+t == 1.0f && s >= 0.0f && t >= 0.0f)
        return 0.5f;

    if (s >= 0.0f && t >= 0.0f && s + t <= 1.0f )
        return 1.0f;*/

    return 0;
}

__device__ __forceinline__ float DistancePointFace3D_gpu(float3 p0, float3 p1, float3 p2, float3 p3, float3 n, bool approx = false) {
    float3 center = (p1 + p2 + p3) * (1.0f / 3.0f);
    if (approx) {
        float d0 = sqrt(dot(p0 - center, p0 - center));
        float d1 = sqrt(dot(p0 - p1, p0 - p1));
        float d2 = sqrt(dot(p0 - p2, p0 - p2));
        float d3 = sqrt(dot(p0 - p3, p0 - p3));
        return min(d0, min(d1, min(d2, d3)));
    }

    // a. Project point onto the plane of the triangle
    float3 p1p0 = p0 - p1;
    float dot_prod = dot(p1p0, n);
    float3 proj = p0 - n * dot_prod;

    //p1p2p3
    float3 cross_p1p2p3 = cross(p2 - p1, p3 - p1);
    float area = norm(cross_p1p2p3) / 2.0f;
    if (area < 1.0e-12) {
        return 1.0e32;
    }

    // b. Test if projection is inside the triangle
    float3 C;

    // edge 0 = p1p2
    float3 edge0 = p2 - p1;
    float3 vp0 = proj - p1;
    C = cross(edge0, vp0);
    float w = (norm(C) / 2.0f) / area;
    if (dot(n, C) < 0.0f) {
        // P is on the right side of edge0
        // compute distance point to segment
        float curr_dist;
        float3 base = edge0 * (1.0f / norm(edge0));
        float Dt = dot(base, vp0);
        if (Dt < 0.0f) {
            curr_dist = norm(p0 - p1);
        }
        else if (Dt > norm(edge0)) {
            curr_dist = norm(p0 - p2);
        }
        else {
            curr_dist = norm(p0 - (p1 + base * Dt));
        }
        return curr_dist;
    }

    // edge 1 = p2p3
    float3 edge1 = p3 - p2;
    float3 vp1 = proj - p2;
    C = cross(edge1, vp1);
    float u = (norm(C) / 2.0f) / area;
    if (dot(n, C) < 0.0f) {
        // P is on the right side of edge1
        // compute distance point to segment
        float curr_dist;
        float3 base = edge1 * (1.0f / norm(edge1));
        float Dt = dot(base, vp1);
        if (Dt < 0.0f) {
            curr_dist = norm(p0 - p2);
        }
        else if (Dt > norm(edge1)) {
            curr_dist = norm(p0 - p3);
        }
        else {
            curr_dist = norm(p0 - (p2 + base * Dt));
        }
        return curr_dist;
    }

    // edge 2 = p3p1
    float3 edge2 = p1 - p3;
    float3 vp2 = proj - p3;
    C = cross(edge2, vp2);
    float v = (norm(C) / 2.0f) / area;
    if (dot(n, C) < 0.0f) {
        // P is on the right side of edge 2;
        float curr_dist;
        float3 base = edge2 * (1.0f / norm(edge2));
        float Dt = dot(base, vp2);
        if (Dt < 0.0f) {
            curr_dist = norm(p0 - p3);
        }
        else if (Dt > norm(edge2)) {
            curr_dist = norm(p0 - p1);
        }
        else {
            curr_dist = norm(p0 - (p3 + base * Dt));
        }
        return curr_dist;
    }

    if (u <= 1.00001f && v <= 1.00001f && w <= 1.00001f) {
        return sqrt(dot(p0 - proj, p0 - proj));
    }
    else {
        return 1.0e32;
    }

    return 1.0e32;
}

__device__ __forceinline__ void LevelSetSemProcess(float* volume, int* volume_l, float* vertices, int* labels, int* faces, float* normals, int nb_faces, int3 size_grid, float3 center_grid, float res_x, float res_y, float res_z, float disp)
{
    unsigned int i = threadIdx.x + blockIdx.x * THREAD_SIZE_X; // cols
    unsigned int j = threadIdx.y + blockIdx.y * THREAD_SIZE_Y; // rows
    unsigned int k = threadIdx.z + blockIdx.z * THREAD_SIZE_Z; // rows
    unsigned int idx = i * size_grid.y * size_grid.z + j * size_grid.z + k;

    if (i > size_grid.x - 1 || j > size_grid.y - 1 || k > size_grid.z - 1)
        return;

    // Get the 3D coordinate
    float3 p0;
    p0.x = (float(i) - float(size_grid.x) / 2.0f) * res_x + center_grid.x;
    p0.y = (float(j) - float(size_grid.y) / 2.0f) * res_y + center_grid.y;
    p0.z = (float(k) - float(size_grid.z) / 2.0f) * res_z + center_grid.z;

    float3 ray1 = make_ray(0.0f, 0.0f, 1.0f);
    float3 ray2 = make_ray(0.0f, 1.0f, 0.0f);
    float3 ray3 = make_ray(1.0f, 0.0f, 0.0f);
    float3 ray4 = make_ray(1.0f, 0.0f, 1.0f);
    float3 ray5 = make_ray(0.0f, 1.0f, 1.0f);
    float3 ray6 = make_ray(1.0f, 1.0f, 0.0f);
    float3 ray7 = make_ray(1.0f, 1.0f, 1.0f);

    // Compute the smallest distance to the faces
    float min_dist = 1.0e32f;
    float sdf = 1.0f;
    int lbl = 0;
    int intersections1 = 0;
    int intersections2 = 0;
    int intersections3 = 0;
    int intersections4 = 0;
    int intersections5 = 0;
    int intersections6 = 0;
    int intersections7 = 0;

    for (int f = 0; f < nb_faces; f++) {
        // Compute distance point to face
        float3 n = make_float3(normals[3 * f], normals[3 * f + 1], normals[3 * f + 2]);

        float3 p1 = make_float3(vertices[3 * faces[3 * f + 0]], vertices[3 * faces[3 * f + 0] + 1], vertices[3 * faces[3 * f + 0] + 2]);
        float3 p2 = make_float3(vertices[3 * faces[3 * f + 1]], vertices[3 * faces[3 * f + 1] + 1], vertices[3 * faces[3 * f + 1] + 2]);
        float3 p3 = make_float3(vertices[3 * faces[3 * f + 2]], vertices[3 * faces[3 * f + 2] + 1], vertices[3 * faces[3 * f + 2] + 2]);

        // Compute line plane intersection
        intersections1 += IsInterectingRayTriangle3D_gpu(ray1, p0, p1, p2, p3, n);
        intersections2 += IsInterectingRayTriangle3D_gpu(ray2, p0, p1, p2, p3, n);
        intersections3 += IsInterectingRayTriangle3D_gpu(ray3, p0, p1, p2, p3, n);
        intersections4 += IsInterectingRayTriangle3D_gpu(ray4, p0, p1, p2, p3, n);
        intersections5 += IsInterectingRayTriangle3D_gpu(ray5, p0, p1, p2, p3, n);
        intersections6 += IsInterectingRayTriangle3D_gpu(ray6, p0, p1, p2, p3, n);
        intersections7 += IsInterectingRayTriangle3D_gpu(ray7, p0, p1, p2, p3, n);

        // Compute point to face distance
        float curr_dist = DistancePointFace3D_gpu(p0, p1, p2, p3, n);

        if((curr_dist < min_dist) || fabs(curr_dist - min_dist) < 10e-6)
        {
            min_dist = curr_dist;
            sdf = curr_dist;
            lbl = labels[faces[3 * f]];
        }
                        
    }

    int countOut = 0;

    countOut += intersections1 % 2 == 0 ? 1 : 0;
    countOut += intersections2 % 2 == 0 ? 1 : 0;  
    countOut += intersections3 % 2 == 0 ? 1 : 0;
    countOut += intersections4 % 2 == 0 ? 1 : 0;
    countOut += intersections5 % 2 == 0 ? 1 : 0;
    countOut += intersections6 % 2 == 0 ? 1 : 0;
    countOut += intersections7 % 2 == 0 ? 1 : 0;

    if (countOut >= 4)
        volume[idx] = sdf;
    else
        volume[idx] = - sdf;

    volume_l[idx] = lbl;

}

__global__ void LevelSetSemKernel(float* volume, int* volume_l, float* vertices, int* labels, int* faces, float* normals, int nb_faces, int3 size_grid, float3 center_grid, float res_x, float res_y, float res_z, float disp)
{
	LevelSetSemProcess(volume, volume_l, vertices, labels, faces, normals, nb_faces, size_grid, center_grid, res_x, res_y, res_z, disp);
}


//////////////////////////////////////////
///******* Function definitions *********/
//////////////////////////////////////////

pair<float***, int***> LevelSet_gpu(float* vertices, int *labels, int* faces, float* normals, int nb_vertices, int nb_faces, int3 size_grid, float3 center_grid, float res_x, float res_y, float res_z, float disp) {
	// Allocate data
	float*** volume = new float** [size_grid.x];
	for (int i = 0; i < size_grid.x; i++) {
		volume[i] = new float* [size_grid.y];
		for (int j = 0; j < size_grid.y; j++) {
			volume[i][j] = new float[size_grid.z];
			for (int k = 0; k < size_grid.z; k++) {
				volume[i][j][k] = 1.0f;
			}
		}
	}

    int*** volume_l = new int** [size_grid.x];
    for (int i = 0; i < size_grid.x; i++) {
        volume_l[i] = new int* [size_grid.y];
        for (int j = 0; j < size_grid.y; j++) {
            volume_l[i][j] = new int[size_grid.z];
            for (int k = 0; k < size_grid.z; k++) {
                volume_l[i][j][k] = 2;
            }
        }
    }

	float* volume_gpu;
	checkCudaErrors(hipMalloc((void**)&volume_gpu, size_grid.x * size_grid.y * size_grid.z * sizeof(float)));

    int* volume_l_gpu;
    checkCudaErrors(hipMalloc((void**)&volume_l_gpu, size_grid.x * size_grid.y * size_grid.z * sizeof(int)));
    
	for (int i = 0; i < size_grid.x; i++) {
		for (int j = 0; j < size_grid.y; j++) {
			checkCudaErrors(hipMemcpy((void*)&volume_gpu[i * size_grid.y * size_grid.z + j * size_grid.z], (void*)volume[i][j], size_grid.z * sizeof(float), hipMemcpyHostToDevice));
            checkCudaErrors(hipMemset((void*)&volume_l_gpu[i * size_grid.y * size_grid.z + j * size_grid.z], 0, size_grid.z * sizeof(int)));
		}
	}

	float* vertices_gpu;
	checkCudaErrors(hipMalloc((void**)&vertices_gpu, 3 * nb_vertices * sizeof(float)));
	checkCudaErrors(hipMemcpy((void*)vertices_gpu, (void*)vertices, 3 * nb_vertices * sizeof(float), hipMemcpyHostToDevice));

    int* labels_gpu;
    checkCudaErrors(hipMalloc((void**)&labels_gpu,  nb_vertices * sizeof(int)));
    checkCudaErrors(hipMemcpy((void*)labels_gpu, (void*)labels,  nb_vertices * sizeof(int), hipMemcpyHostToDevice));

	int* faces_gpu;
	checkCudaErrors(hipMalloc((void**)&faces_gpu, 3 * nb_faces * sizeof(int)));
	checkCudaErrors(hipMemcpy((void*)faces_gpu, (void*)faces, 3 * nb_faces * sizeof(int), hipMemcpyHostToDevice));

	float* normals_gpu;
	checkCudaErrors(hipMalloc((void**)&normals_gpu, 3 * nb_faces * sizeof(float)));
	checkCudaErrors(hipMemcpy((void*)normals_gpu, (void*)normals, 3 * nb_faces * sizeof(float), hipMemcpyHostToDevice));


	dim3 dimBlock(THREAD_SIZE_X, THREAD_SIZE_Y, THREAD_SIZE_Z);
	dim3 dimGrid(1, 1, 1);
	dimGrid.x = divUp(size_grid.x, dimBlock.x); // #cols
	dimGrid.y = divUp(size_grid.y, dimBlock.y); // # rows
	dimGrid.z = divUp(size_grid.z, dimBlock.z); // # rows

    std::cout << "Start level set on GPU" << std::endl;

	LevelSetSemKernel << <dimGrid, dimBlock >> > (volume_gpu, volume_l_gpu, vertices_gpu, labels_gpu, faces_gpu, normals_gpu, nb_faces, size_grid, center_grid, res_x, res_y, res_z, disp);
   
	checkCudaErrors(hipDeviceSynchronize());

    std::cout << "End level set on GPU" << std::endl;

	for (int i = 0; i < size_grid.x; i++) {
		for (int j = 0; j < size_grid.y; j++) {
			checkCudaErrors(hipMemcpy((void*)volume[i][j], (void*)&volume_gpu[i * size_grid.y * size_grid.z + j * size_grid.z], size_grid.z * sizeof(float), hipMemcpyDeviceToHost));
            checkCudaErrors(hipMemcpy((void*)volume_l[i][j], (void*)&volume_l_gpu[i * size_grid.y * size_grid.z + j * size_grid.z], size_grid.z * sizeof(int), hipMemcpyDeviceToHost));
            
		}
	}

	checkCudaErrors(hipFree(volume_gpu));
	checkCudaErrors(hipFree(vertices_gpu));
    checkCudaErrors(hipFree(labels_gpu));
	checkCudaErrors(hipFree(faces_gpu));
	checkCudaErrors(hipFree(normals_gpu));

	return pair<float***, int***>(volume, volume_l);
}
